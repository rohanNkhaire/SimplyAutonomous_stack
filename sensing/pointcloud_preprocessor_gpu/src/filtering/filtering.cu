#include "hip/hip_runtime.h"
#include "pointcloud_preprocessor_gpu/filtering/kernel.hpp"

__global__ void filter_pointcloud(float* filtered_points, float* points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  float4 point = ((float4*)points)[point_idx];

  if(point.x>min_x_range && point.x<max_x_range
    && point.y>min_y_range && point.y<max_y_range
    && point.z>min_z_range && point.z<max_z_range) return;


  float* address = filtered_points + (point_idx)*4;

  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
 
}

hipError_t generate_filtered_pointcloud(float* filtered_points, float* points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        hipStream_t stream)
{
	int threadNum = 256;
  dim3 blocks((points_size+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  filter_pointcloud<<<blocks, threads, 0, stream>>>
    (filtered_points, points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range);

  hipError_t err = hipGetLastError();

  return err;
}