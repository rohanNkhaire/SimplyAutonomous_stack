#include "hip/hip_runtime.h"
#include "pointcloud_preprocessor_gpu/filtering/kernel.hpp"

__global__ void filter_pointcloud(int* filtered_points, int* points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  int4 point = ((int4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range) return;

	filtered_points[point_idx+0] = point.x;
	filtered_points[point_idx+1] = point.y;
	filtered_points[point_idx+2] = point.z;
	filtered_points[point_idx+3] = point.w;
	
}

hipError_t generate_filtered_pointcloud(int* filtered_points, int* points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        hipStream_t stream = 0)
{
	int threadNum = 256;
  dim3 blocks((points_size+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  filter_pointcloud<<<blocks, threads, 0, stream>>>
    (filtered_points, points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range);
  hipError_t err = hipGetLastError();

  return err;
}